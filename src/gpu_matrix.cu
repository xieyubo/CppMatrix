#include "hip/hip_runtime.h"
#include "std_patch.h"
#include <hip/hip_runtime.h>
#include <memory>
#include <stdexcept>

__global__ void vectorPow(const float* A, const float* B, float* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = powf(A[i], B[i]);
    }
}

void CudaFree(void* p)
{
    if (p) {
        auto err = hipFree(p);
        if (err != hipSuccess) {
            throw std::runtime_error { "hipFree failed." };
        }
    }
}

template <typename T>
std::unique_ptr<T, decltype(&CudaFree)> CudaMalloc(size_t numElements)
{
    T* p {};
    auto err = hipMalloc((void**)&p, sizeof(T) * numElements);
    if (err != hipSuccess) {
        throw std::runtime_error { "hipMalloc failed." };
    }
    return { p, &CudaFree };
}

void CudaPow(std::float16_t* inputA, std::float16_t* inputB, std::float16_t* output, size_t bufferSize)
{
    throw std::runtime_error { "Not implemented." };
}

void CudaPow(std::float32_t* inputA, std::float32_t* inputB, std::float32_t* output, size_t bufferSize)
{
    auto numElements = bufferSize / sizeof(std::float32_t);

    // Allocate the device input vector A.
    auto d_A = CudaMalloc<std::float32_t>(numElements);

    // Allocate the device input vector B.
    auto d_B = CudaMalloc<std::float32_t>(numElements);

    // Allocate the output vector.
    auto d_C = CudaMalloc<std::float32_t>(numElements);

    // Copy host to device.
    auto err = hipMemcpy(d_A.get(), inputA, bufferSize, hipMemcpyHostToDevice);
    err = hipMemcpy(d_B.get(), inputB, bufferSize, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorPow<<<blocksPerGrid, threadsPerBlock>>>(d_A.get(), d_B.get(), d_C.get(), numElements);
    err = hipGetLastError();

    // Copy from device to host.
    err = hipMemcpy(output, d_C.get(), bufferSize, hipMemcpyDeviceToHost);
}